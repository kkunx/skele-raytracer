#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <iostream>
#include <ostream>
#include <sstream>
#include <time.h>

#include "raytrace.h"
#include "vec3.h"

void generate_rays(Scene scene, Options option, char *output);


__global__ void ray_generation(vecmath::vec3 *image, Scene **scene)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if(x >= 1920 || y >= 1080)
	{
		return;
	}

	// the pixel index is y * 1920 since 1920 is the max x value, and then this gets us to the y coordinate
	// and then shift over by x amount to hit pixel (x, y)
	int pixel = y * 1920 + x;

	image[pixel] = vecmath::vec3((float) x / 1920.0f, (float) y / 1920.0f, 0.1f);
}


void generate_rays(Scene scene, Options option, char *output)
{
	// The output image that will be written to
	size_t image_size = scene.width * scene.height * sizeof(vecmath::vec3);

	vecmath::vec3 *image_host = (vecmath::vec3*) malloc(image_size * sizeof(vecmath::vec3));

	vecmath::vec3 *image;
	hipMalloc((void**) &image, image_size);

	Scene **cuda_scene_data;
	hipMalloc((void**) &cuda_scene_data, scene.size() * sizeof(Scene*));
	printf("sizeof scene: %lu\n", sizeof(scene));
	printf("Size of this scene: %lu\n", scene.size());

	int thread_x = 8;
	int thread_y = 8;

	dim3 blocks;
	blocks.x = scene.width / thread_x + 1;
	blocks.y = scene.height / thread_y + 1;
	blocks.z = 1;

	dim3 grid;
	grid.x = thread_x;
	grid.y = thread_y;
	grid.z = 1;

	hipMemcpy(image, image_host, image_size, hipMemcpyHostToDevice);

	ray_generation<<<blocks, grid>>>(image, cuda_scene_data);
	printf("Got out\n");
	hipMemcpy(image_host, image, image_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();



	// Read back on the host
	std::ofstream ofs(output, std::ios::out | std::ios::binary);
	ofs << "P6\n"
		<< scene.width << " " << scene.height << "\n255\n";

	for(int i = 0; i < scene.height; i++)
	{
		for(int j = 0; j < scene.width; j++)
		{
			int index = i * scene.width + j;
			printf("pixel[%d]: %f %f %f\n", index, image_host[index].x, image_host[index].y, image_host[index].z);
			ofs << (unsigned char) (std::min(float(1), image_host[index].x) * 255) << (unsigned char) (std::min(float(1), image_host[index].y) * 255) << (unsigned char) (std::min(float(1), image_host[index].z) * 255);

		}
	}
	ofs.close();
	printf("***\nWROTE TO PPM\n***\n");

}

/*

	///////////////// Code for ray generation, can call this kernel "main_renderer" or something
	// Code for without a grid_size operates similarily without r.
	if(option.grid_size > 0)
	{
		for(int i = 0; i < option.grid_size; i++)
		{
			for(int j = 0; j < option.grid_size; j++)
			{
				// r adds some jitter to the ray that we're going to cast
				// u and v are basically the x / y coordinates transformed by the angle (fov basically) and the screen's aspect ratio
				float r = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
				float u = (2 * ((x + r) * inv_width) - 1) * angle * aspect_ratio;
				float v = (1 - 2 * ((y + r) * inv_height)) * angle;

				// Create the ray's direction vector as a combination of where our camera is looking, and the u & v pixel offsets
				// This will cast out to all pixels in the screen as these loops iterate and construct new u's and v's
				vecmath::vec3 ray_dir(scene.camera.direction + u * scene.camera.right + v * scene.camera.up);
				vecmath::normalize(ray_dir);

				Ray ray;
				ray.position  = scene.camera.position;
				ray.direction = ray_dir;

				// Iteratively add the results to shade for each grid computation to [y][x] in image
				image[y][x] += shade(ray, scene, option.max_depth, option.monte_carlo, option.num_path_traces);
			}
		}
		image[y][x] /= (option.grid_size * option.grid_size);
	}

	else
	{
		float u = (2 * ((x + 0.5) * inv_width) - 1) * angle * aspect_ratio;
		float v = (1 - 2 * ((y + 0.5) * inv_height)) * angle;

		vecmath::vec3 ray_dir(scene.camera.direction + u * scene.camera.right + v * scene.camera.up);
		vecmath::normalize(ray_dir);

		Ray ray;
		ray.position  = scene.camera.position;
		ray.direction = ray_dir;

		// Output the results of shade to the image at index [y][x]
		image[y][x] = shade(ray, scene, option.max_depth, option.monte_carlo, option.num_path_traces);
	}

	// Definitions for what ray corresponds to what pixel
	float inv_width	   = 1 / float(scene.width);
	float inv_height   = 1 / float(scene.height);
	float aspect_ratio = scene.width / float(scene.height);
	float angle		   = tan(M_PI * 0.5 * option.fov / 180.0f);



	std::ofstream ofs(output, std::ios::out | std::ios::binary);
	ofs << "P6\n"
		<< scene.width << " " << scene.height << "\n255\n";

	for(int i = 0; i < scene.height; i++)
	{
		for(int j = 0; j < scene.width; j++)
		{
			ofs << (unsigned char) (std::min(float(1), image[i][j].x) * 255) << (unsigned char) (std::min(float(1), image[i][j].y) * 255) << (unsigned char) (std::min(float(1), image[i][j].z) * 255);
		}
	}

	ofs.close();

	printf("***\nWROTE TO PPM\n***\n");

	delete[] image;
}*/


int main(int argc, char *argv[])
{

	Options option;
	Scene scene;

	int width  = scene.width;
	int height = scene.height;

	char *path;
	char *output;

	bool output_path_passed = false;
	bool path_passed		= false;
	bool use_shadows;

	for(int i = 0; i < argc; i++)
	{
		if(strcmp(argv[i], "--gillum") == 0)
		{
			if(i + 1 < argc)
			{
				option.monte_carlo	   = true;
				option.num_path_traces = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "gillum takes an int after flag for the number of paths traced" << std::endl;
			}
		}

		if(strcmp(argv[i], "--fov") == 0)
		{
			if(i + 1 < argc)
			{
				option.fov = atof(argv[i + 1]);
			}

			else
			{
				std::cerr << "fov takes a float (degrees) after flag for the field of view" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--jsample") == 0)
		{
			if(i + 1 < argc)
			{
				option.grid_size = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "jsample takes an int after flag for the supersampling grid size" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--width") == 0)
		{
			if(i + 1 < argc)
			{
				width = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "width takes an int after flag for the width" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--height") == 0)
		{
			if(i + 1 < argc)
			{
				height = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "height takes an int after flag for the width" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--depth") == 0)
		{
			if(i + 1 < argc && atoi(argv[i + 1]) > 0)
			{
				option.max_depth = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "depth takes a positive int after flag for the max depth" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--parallel") == 0)
		{
			if(i + 1 < argc && strcmp(argv[i + 1], "true") == 0)
			{
				option.visual = false;
			}

			if(i + 1 < argc && strcmp(argv[i + 1], "false") == 0)
			{
				option.visual = true;
			}
		}

		if(strcmp(argv[i], "--path") == 0)
		{
			if(i + 1 < argc)
			{
				path		= argv[i + 1];
				path_passed = true;
			}

			else
			{
				std::cerr << "path must be passed after --path" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--output") == 0)
		{
			if(i + 1 < argc)
			{
				output			   = argv[i + 1];
				output_path_passed = true;
			}

			else
			{
				std::cerr << "output path must be passed after --output" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--shadow") == 0)
		{
			use_shadows = true;
		}
	}

	if(!path_passed)
	{
		std::cerr << "no scene file was passed. Pass with --path path_to_scn" << std::endl;
		return 0;
	}

	if(!output_path_passed)
	{
		std::cerr << "no output destination was passed. Pass with --output destination_path.ppm" << std::endl;
		return 0;
	}

	scene			  = parseScene(path);
	scene.width		  = width;
	scene.height	  = height;
	scene.use_shadows = use_shadows;

	option.to_string();

	srand((unsigned) time(0));


	generate_rays(scene, option, output);




	return 0;
}
